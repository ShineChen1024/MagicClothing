#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include <vector>

#include "utils/checks.h"
#include "utils/cuda.cuh"
#include "inplace_abn.h"

#include <ATen/cuda/HIPContext.h>

// Operations for reduce
struct SumOpH {
  __device__ SumOpH(const half *t, int c, int s)
      : tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    return __half2float(tensor[(batch * chn + plane) * sp + n]);
  }
  const half *tensor;
  const int chn;
  const int sp;
};

struct VarOpH {
  __device__ VarOpH(float m, const half *t, int c, int s)
      : mean(m), tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    const auto t = __half2float(tensor[(batch * chn + plane) * sp + n]);
    return (t - mean) * (t - mean);
  }
  const float mean;
  const half *tensor;
  const int chn;
  const int sp;
};

struct GradOpH {
  __device__ GradOpH(float _weight, float _bias, const half *_z, const half *_dz, int c, int s)
      : weight(_weight), bias(_bias), z(_z), dz(_dz), chn(c), sp(s) {}
  __device__ __forceinline__ Pair<float> operator()(int batch, int plane, int n) {
    float _y = (__half2float(z[(batch * chn + plane) * sp + n]) - bias) / weight;
    float _dz = __half2float(dz[(batch * chn + plane) * sp + n]);
    return Pair<float>(_dz, _y * _dz);
  }
  const float weight;
  const float bias;
  const half *z;
  const half *dz;
  const int chn;
  const int sp;
};

/***********
 * mean_var
 ***********/

__global__ void mean_var_kernel_h(const half *x, float *mean, float *var, int num, int chn, int sp) {
  int plane = blockIdx.x;
  float norm = 1.f / static_cast<float>(num * sp);

  float _mean = reduce<float, SumOpH>(SumOpH(x, chn, sp), plane, num, sp) * norm;
  __syncthreads();
  float _var = reduce<float, VarOpH>(VarOpH(_mean, x, chn, sp), plane, num, sp) * norm;

  if (threadIdx.x == 0) {
    mean[plane] = _mean;
    var[plane] = _var;
  }
}

std::vector<at::Tensor> mean_var_cuda_h(at::Tensor x) {
  CHECK_CUDA_INPUT(x);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Prepare output tensors
  auto mean = at::empty({chn},x.options().dtype(at::kFloat));
  auto var = at::empty({chn},x.options().dtype(at::kFloat));

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  mean_var_kernel_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<half*>(x.data<at::Half>()),
      mean.data<float>(),
      var.data<float>(),
      num, chn, sp);

  return {mean, var};
}

/**********
 * forward
 **********/

__global__ void forward_kernel_h(half *x, const float *mean, const float *var, const float *weight, const float *bias,
                                 bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  const float _mean = mean[plane];
  const float _var = var[plane];
  const float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  const float _bias = affine ? bias[plane] : 0.f;

  const float mul = rsqrt(_var + eps) * _weight;

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      half *x_ptr = x + (batch * chn + plane) * sp + n;
      float _x = __half2float(*x_ptr);
      float _y = (_x - _mean) * mul + _bias;

      *x_ptr = __float2half(_y);
    }
  }
}

at::Tensor forward_cuda_h(at::Tensor x, at::Tensor mean, at::Tensor var, at::Tensor weight, at::Tensor bias,
                        bool affine, float eps) {
  CHECK_CUDA_INPUT(x);
  CHECK_CUDA_INPUT(mean);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  forward_kernel_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<half*>(x.data<at::Half>()),
      mean.data<float>(),
      var.data<float>(),
      weight.data<float>(),
      bias.data<float>(),
      affine, eps, num, chn, sp);

  return x;
}

__global__ void edz_eydz_kernel_h(const half *z, const half *dz, const float *weight, const float *bias,
                                float *edz, float *eydz, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;

  Pair<float> res = reduce<Pair<float>, GradOpH>(GradOpH(_weight, _bias, z, dz, chn, sp), plane, num, sp);
  __syncthreads();

  if (threadIdx.x == 0) {
    edz[plane] = res.v1;
    eydz[plane] = res.v2;
  }
}

std::vector<at::Tensor> edz_eydz_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor weight, at::Tensor bias,
                                      bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto edz = at::empty({chn},z.options().dtype(at::kFloat));
  auto eydz = at::empty({chn},z.options().dtype(at::kFloat));

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  edz_eydz_kernel_h<<<blocks, threads, 0, stream>>>(
        reinterpret_cast<half*>(z.data<at::Half>()),
        reinterpret_cast<half*>(dz.data<at::Half>()),
        weight.data<float>(),
        bias.data<float>(),
        edz.data<float>(),
        eydz.data<float>(),
        affine, eps, num, chn, sp);
 
  return {edz, eydz};
}

__global__ void backward_kernel_h(const half *z, const half *dz, const float *var, const float *weight, const float *bias, const float *edz,
                                  const float *eydz, half *dx, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;
  float _var = var[plane];
  float _edz = edz[plane];
  float _eydz = eydz[plane];

  float _mul = _weight * rsqrt(_var + eps);
  float count = float(num * sp);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      float _dz = __half2float(dz[(batch * chn + plane) * sp + n]);
      float _y = (__half2float(z[(batch * chn + plane) * sp + n]) - _bias) / _weight;

      dx[(batch * chn + plane) * sp + n] = __float2half((_dz - _edz / count - _y * _eydz / count) * _mul);
    }
  }
}

at::Tensor backward_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor var, at::Tensor weight, at::Tensor bias,
                                      at::Tensor edz, at::Tensor eydz, bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);
  CHECK_CUDA_INPUT(edz);
  CHECK_CUDA_INPUT(eydz);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto dx = at::zeros_like(z);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  backward_kernel_h<<<blocks, threads, 0, stream>>>(
        reinterpret_cast<half*>(z.data<at::Half>()),
        reinterpret_cast<half*>(dz.data<at::Half>()),
        var.data<float>(),
        weight.data<float>(),
        bias.data<float>(),
        edz.data<float>(),
        eydz.data<float>(),
        reinterpret_cast<half*>(dx.data<at::Half>()),
        affine, eps, num, chn, sp);

  return dx;
}

__global__ void leaky_relu_backward_impl_h(half *z, half *dz, float slope, int64_t count) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count;  i += blockDim.x * gridDim.x){
    float _z = __half2float(z[i]);
    if (_z < 0) {
      dz[i] = __float2half(__half2float(dz[i]) * slope);
      z[i] = __float2half(_z / slope);
    }
  }
}

void leaky_relu_backward_cuda_h(at::Tensor z, at::Tensor dz, float slope) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);

  int64_t count = z.numel();
  dim3 threads(getNumThreads(count));
  dim3 blocks = (count + threads.x - 1) / threads.x;
  auto stream = at::cuda::getCurrentCUDAStream();
  leaky_relu_backward_impl_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<half*>(z.data<at::Half>()),
      reinterpret_cast<half*>(dz.data<at::Half>()),
      slope, count);
}

