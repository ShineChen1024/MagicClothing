#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <vector>

#include "utils/checks.h"
#include "utils/cuda.cuh"
#include "inplace_abn.h"

#include <ATen/cuda/HIPContext.h>

// Operations for reduce
template<typename T>
struct SumOp {
  __device__ SumOp(const T *t, int c, int s)
      : tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ T operator()(int batch, int plane, int n) {
    return tensor[(batch * chn + plane) * sp + n];
  }
  const T *tensor;
  const int chn;
  const int sp;
};

template<typename T>
struct VarOp {
  __device__ VarOp(T m, const T *t, int c, int s)
      : mean(m), tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ T operator()(int batch, int plane, int n) {
    T val = tensor[(batch * chn + plane) * sp + n];
    return (val - mean) * (val - mean);
  }
  const T mean;
  const T *tensor;
  const int chn;
  const int sp;
};

template<typename T>
struct GradOp {
  __device__ GradOp(T _weight, T _bias, const T *_z, const T *_dz, int c, int s)
      : weight(_weight), bias(_bias), z(_z), dz(_dz), chn(c), sp(s) {}
  __device__ __forceinline__ Pair<T> operator()(int batch, int plane, int n) {
    T _y = (z[(batch * chn + plane) * sp + n] - bias) / weight;
    T _dz = dz[(batch * chn + plane) * sp + n];
    return Pair<T>(_dz, _y * _dz);
  }
  const T weight;
  const T bias;
  const T *z;
  const T *dz;
  const int chn;
  const int sp;
};

/***********
 * mean_var
 ***********/

template<typename T>
__global__ void mean_var_kernel(const T *x, T *mean, T *var, int num, int chn, int sp) {
  int plane = blockIdx.x;
  T norm = T(1) / T(num * sp);

  T _mean = reduce<T, SumOp<T>>(SumOp<T>(x, chn, sp), plane, num, sp) * norm;
  __syncthreads();
  T _var = reduce<T, VarOp<T>>(VarOp<T>(_mean, x, chn, sp), plane, num, sp) * norm;

  if (threadIdx.x == 0) {
    mean[plane] = _mean;
    var[plane] = _var;
  }
}

std::vector<at::Tensor> mean_var_cuda(at::Tensor x) {
  CHECK_CUDA_INPUT(x);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Prepare output tensors
  auto mean = at::empty({chn}, x.options());
  auto var = at::empty({chn}, x.options());

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.type(), "mean_var_cuda", ([&] {
    mean_var_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        x.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        num, chn, sp);
  }));

  return {mean, var};
}

/**********
 * forward
 **********/

template<typename T>
__global__ void forward_kernel(T *x, const T *mean, const T *var, const T *weight, const T *bias,
                               bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _mean = mean[plane];
  T _var = var[plane];
  T _weight = affine ? abs(weight[plane]) + eps : T(1);
  T _bias = affine ? bias[plane] : T(0);

  T mul = rsqrt(_var + eps) * _weight;

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      T _x = x[(batch * chn + plane) * sp + n];
      T _y = (_x - _mean) * mul + _bias;

      x[(batch * chn + plane) * sp + n] = _y;
    }
  }
}

at::Tensor forward_cuda(at::Tensor x, at::Tensor mean, at::Tensor var, at::Tensor weight, at::Tensor bias,
                        bool affine, float eps) {
  CHECK_CUDA_INPUT(x);
  CHECK_CUDA_INPUT(mean);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.type(), "forward_cuda", ([&] {
    forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        x.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return x;
}

/***********
 * edz_eydz
 ***********/

template<typename T>
__global__ void edz_eydz_kernel(const T *z, const T *dz, const T *weight, const T *bias,
                                T *edz, T *eydz, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _weight = affine ? abs(weight[plane]) + eps : 1.f;
  T _bias = affine ? bias[plane] : 0.f;

  Pair<T> res = reduce<Pair<T>, GradOp<T>>(GradOp<T>(_weight, _bias, z, dz, chn, sp), plane, num, sp);
  __syncthreads();

  if (threadIdx.x == 0) {
    edz[plane] = res.v1;
    eydz[plane] = res.v2;
  }
}

std::vector<at::Tensor> edz_eydz_cuda(at::Tensor z, at::Tensor dz, at::Tensor weight, at::Tensor bias,
                                      bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto edz = at::empty({chn}, z.options());
  auto eydz = at::empty({chn}, z.options());

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(z.type(), "edz_eydz_cuda", ([&] {
    edz_eydz_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        z.data<scalar_t>(),
        dz.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        edz.data<scalar_t>(),
        eydz.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return {edz, eydz};
}

/***********
 * backward
 ***********/

template<typename T>
__global__ void backward_kernel(const T *z, const T *dz, const T *var, const T *weight, const T *bias, const T *edz,
	                        const T *eydz, T *dx, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _weight = affine ? abs(weight[plane]) + eps : 1.f;
  T _bias = affine ? bias[plane] : 0.f;
  T _var = var[plane];
  T _edz = edz[plane];
  T _eydz = eydz[plane];

  T _mul = _weight * rsqrt(_var + eps);
  T count = T(num * sp);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      T _dz = dz[(batch * chn + plane) * sp + n];
      T _y = (z[(batch * chn + plane) * sp + n] - _bias) / _weight;

      dx[(batch * chn + plane) * sp + n] = (_dz - _edz / count - _y * _eydz / count) * _mul;
    }
  }
}

at::Tensor backward_cuda(at::Tensor z, at::Tensor dz, at::Tensor var, at::Tensor weight, at::Tensor bias,
                                      at::Tensor edz, at::Tensor eydz, bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);
  CHECK_CUDA_INPUT(edz);
  CHECK_CUDA_INPUT(eydz);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto dx = at::zeros_like(z);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(z.type(), "backward_cuda", ([&] {
    backward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        z.data<scalar_t>(),
        dz.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        edz.data<scalar_t>(),
        eydz.data<scalar_t>(),
        dx.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return dx;
}

/**************
 * activations
 **************/

template<typename T>
inline void leaky_relu_backward_impl(T *z, T *dz, float slope, int64_t count) {
  // Create thrust pointers
  thrust::device_ptr<T> th_z = thrust::device_pointer_cast(z);
  thrust::device_ptr<T> th_dz = thrust::device_pointer_cast(dz);

  auto stream = at::cuda::getCurrentCUDAStream();
  thrust::transform_if(thrust::cuda::par.on(stream),
                       th_dz, th_dz + count, th_z, th_dz,
                       [slope] __device__ (const T& dz) { return dz * slope; },
                       [] __device__ (const T& z) { return z < 0; });
  thrust::transform_if(thrust::cuda::par.on(stream),
                       th_z, th_z + count, th_z,
                       [slope] __device__ (const T& z) { return z / slope; },
                       [] __device__ (const T& z) { return z < 0; });
}

void leaky_relu_backward_cuda(at::Tensor z, at::Tensor dz, float slope) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);

  int64_t count = z.numel();

  AT_DISPATCH_FLOATING_TYPES(z.type(), "leaky_relu_backward_cuda", ([&] {
    leaky_relu_backward_impl<scalar_t>(z.data<scalar_t>(), dz.data<scalar_t>(), slope, count);
  }));
}

template<typename T>
inline void elu_backward_impl(T *z, T *dz, int64_t count) {
  // Create thrust pointers
  thrust::device_ptr<T> th_z = thrust::device_pointer_cast(z);
  thrust::device_ptr<T> th_dz = thrust::device_pointer_cast(dz);

  auto stream = at::cuda::getCurrentCUDAStream();
  thrust::transform_if(thrust::cuda::par.on(stream),
                       th_dz, th_dz + count, th_z, th_z, th_dz,
                       [] __device__ (const T& dz, const T& z) { return dz * (z + 1.); },
                       [] __device__ (const T& z) { return z < 0; });
  thrust::transform_if(thrust::cuda::par.on(stream),
                       th_z, th_z + count, th_z,
                       [] __device__ (const T& z) { return log1p(z); },
                       [] __device__ (const T& z) { return z < 0; });
}

void elu_backward_cuda(at::Tensor z, at::Tensor dz) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);

  int64_t count = z.numel();

  AT_DISPATCH_FLOATING_TYPES(z.type(), "leaky_relu_backward_cuda", ([&] {
    elu_backward_impl<scalar_t>(z.data<scalar_t>(), dz.data<scalar_t>(), count);
  }));
}
