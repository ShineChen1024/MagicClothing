// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace detectron2 {
int get_cudart_version() {
  return CUDART_VERSION;
}
} // namespace detectron2
